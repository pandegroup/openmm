
#include <hip/hip_runtime.h>
/**
 * This file contains CUDA definitions for the macros and functions needed for the
 * common compute framework.
 */

#define KERNEL extern "C" __global__
#define LOCAL __shared__
#define GLOBAL
#define RESTRICT __restrict__
#define LOCAL_ID threadIdx.x
#define LOCAL_SIZE blockDim.x
#define GLOBAL_ID blockIdx.x*blockDim.x+threadIdx.x
#define GLOBAL_SiZE blockDim.x*gridDim.x
#define GROUP_ID blockIdx.x
#define NUM_GROUPS gridDim.x
#define SYNC_THREADS __syncthreads();
